#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void Draw(char *frame) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	int idx;
	//printf("y: %d", y);
	//printf("x: %d", x);
	if (y < H & x < W) {
		char c;
		
		if (x == W-1) {
			c = y == H-1 ? '\0' : '\n';
		}
		else if (y == 0 | y == H-1 | x == 0 | x == W-2) {
			c = ':';
		} 
		else if(y>=5 & y<=10){
			if (x==34 & y==10){
				c='#';
			}
			else if (x==34){
				c='|';
			}
			else{
				switch(y){
					case 5:
						if(x >18 & x<=22){
							c='#';
						}
						else if(x==33){
							c = '<';
						}
						else{
							c=' ';
						}
						break;
					case 6:
						if(x >16 & x<=22){
							c='#';
						}
						else{
							c=' ';
						}
						break;
					case 7:
						if(x >14 & x<=22){
							c='#';
						}
						else{
							c=' ';
						}
						break;
					case 8:
						if(x >12 & x<=22){
							c='#';
						}
						else{
							c=' ';
						}
						break;
					case 9:
						if(x >10 & x<=22){
							c='#';
						}
						else{
							c=' ';
						}
						break;
					case 10:
						if(x >8 & x<=22){
							c='#';
						}
						else{
							c=' ';
						}
						break;
				}
			}
		} 
		else {
			c = ' ';
		}
		
		frame[y*W+x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	system("pause");
	return 0;
}
